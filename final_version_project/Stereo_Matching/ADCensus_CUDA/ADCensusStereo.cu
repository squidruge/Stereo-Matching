#include "hip/hip_runtime.h"
﻿#include "ADCensusStereo.h"
#include <algorithm>
#include <chrono>
using namespace std::chrono;

ADCensusStereo::ADCensusStereo() : width_(0), height_(0), img_left_(nullptr), img_right_(nullptr),
disp_left_(nullptr), disp_right_(nullptr),
is_initialized_(false) { }

ADCensusStereo::~ADCensusStereo()
{
	Release();
	is_initialized_ = false;
}

bool ADCensusStereo::Initialize(const sint32& width, const sint32& height, const ADCensusOption& option)
{
	// ··· 赋值

	// 影像尺寸
	width_ = width;
	height_ = height;
	// 算法参数
	option_ = option;

	if (width <= 0 || height <= 0) {
		return false;
	}

	//··· 开辟内存空间
	const sint32 img_size = width_ * height_;
	const sint32 disp_range = option_.max_disparity - option_.min_disparity;
	if (disp_range <= 0) {
		return false;
	}

	// 视差图
	disp_left_ = new float32[img_size];
	disp_right_ = new float32[img_size];

	// 初始化代价计算器
	if (!cost_computer_.Initialize(width_, height_, option_.min_disparity, option_.max_disparity)) {
		is_initialized_ = false;
		return is_initialized_;
	}

	// 初始化代价聚合器
	if (!aggregator_.Initialize(width_, height_, option_.min_disparity, option_.max_disparity)) {
		is_initialized_ = false;
		return is_initialized_;
	}

	// 初始化多步优化器
	if (!refiner_.Initialize(width_, height_)) {
		is_initialized_ = false;
		return is_initialized_;
	}

	is_initialized_ = disp_left_ && disp_right_;

	return is_initialized_;
}

bool ADCensusStereo::Match(const uint8* img_left, const uint8* img_right, float32* disp_left)
{
	if (!is_initialized_) {
		return false;
	}
	if (img_left == nullptr || img_right == nullptr || disp_left == nullptr) {
		return false;
	}

	img_left_ = img_left;
	img_right_ = img_right;

	auto start = steady_clock::now();

	// 代价计算
	ComputeCost();

	auto end = steady_clock::now();
	auto tt = duration_cast<milliseconds>(end - start);
#if LOG
	printf("代价计算耗时 :	%lf s\n", tt.count() / 1000.0);
#endif // LOG

	start = steady_clock::now();

	// 代价聚合
	CostAggregation();

	end = steady_clock::now();
	tt = duration_cast<milliseconds>(end - start);
#if LOG
	printf("代价聚合耗时 :	%lf s\n", tt.count() / 1000.0);
#endif // LOG


	start = steady_clock::now();

	// 扫描线优化
	ScanlineOptimize();

	end = steady_clock::now();
	tt = duration_cast<milliseconds>(end - start);

#if LOG
	printf("扫描线优化耗时 :	%lf s\n", tt.count() / 1000.0);
#endif // LOG
	start = steady_clock::now();

	// 计算左右视图视差
	ComputeDisparity();
	ComputeDisparityRight();

	end = steady_clock::now();
	tt = duration_cast<milliseconds>(end - start);
#if LOG
	printf("视差计算耗时 :	%lf s\n", tt.count() / 1000.0);
#endif // LOG

	start = steady_clock::now();

	// 多步骤视差优化
	MultiStepRefine();

	end = steady_clock::now();
	tt = duration_cast<milliseconds>(end - start);
#if LOG
	printf("视差优化耗时 :	%lf s\n", tt.count() / 1000.0);
#endif // LOG

	start = steady_clock::now();

	// 输出视差图
	memcpy(disp_left, disp_left_, height_ * width_ * sizeof(float32));

	end = steady_clock::now();
	tt = duration_cast<milliseconds>(end - start);

#if LOG
	printf("输出视差耗时 :	%lf s\n", tt.count() / 1000.0);
#endif // LOG
	return true;
}

bool ADCensusStereo::Reset(const uint32& width, const uint32& height, const ADCensusOption& option)
{
	// 释放内存
	Release();

	// 重置初始化标记
	is_initialized_ = false;

	// 初始化
	return Initialize(width, height, option);
}


void ADCensusStereo::ComputeCost()
{
	// 设置代价计算器数据
	cost_computer_.SetData(img_left_, img_right_);
	// 设置代价计算器参数
	cost_computer_.SetParams(option_.lambda_ad, option_.lambda_census);
	// 计算代价
	cost_computer_.Compute();
}

void ADCensusStereo::CostAggregation()
{
	// 设置聚合器数据
	aggregator_.SetData(img_left_, img_right_, cost_computer_.get_cost_ptr());
	// 设置聚合器参数
	aggregator_.SetParams(option_.cross_L1, option_.cross_L2, option_.cross_t1, option_.cross_t2);
	// 代价聚合
	aggregator_.Aggregate(4);
}

void ADCensusStereo::ScanlineOptimize()
{
	// 设置优化器数据
	scan_line_.SetData(img_left_, img_right_, cost_computer_.get_cost_ptr(), aggregator_.get_cost_ptr());
	// 设置优化器参数
	scan_line_.SetParam(width_, height_, option_.min_disparity, option_.max_disparity, option_.so_p1, option_.so_p2, option_.so_tso);
	// 扫描线优化
	scan_line_.Optimize();
}

void ADCensusStereo::MultiStepRefine()
{
	// 设置多步优化器数据
	refiner_.SetData(img_left_, aggregator_.get_cost_ptr(), aggregator_.get_arms_ptr(), disp_left_, disp_right_);
	// 设置多步优化器参数
	refiner_.SetParam(option_.min_disparity, option_.max_disparity, option_.irv_ts, option_.irv_th, option_.lrcheck_thres,
		option_.do_lr_check, option_.do_filling, option_.do_filling, option_.do_discontinuity_adjustment);
	// 多步优化
	refiner_.Refine();
}
__global__ void ComputeDisparityCuda(float32* cost_ptr, float32* disparity,
	const sint32 height, const sint32 width, sint32 max_disparity, sint32 min_disparity)
{

	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	auto disp_range = max_disparity - min_disparity;
	if (idx < width && idy < height)
	{
		//float32* cost_local = (float32*)malloc(sizeof(float32) * disp_range);
		float32 min_cost = Large_Float;
		float32 sec_min_cost = Large_Float;
		sint32 best_disparity = 0;

		// ---遍历视差范围内的所有代价值，输出最小代价值及对应的视差值
		for (sint32 d = min_disparity; d < max_disparity; d++) {
			const sint32 d_idx = d - min_disparity;
			//const auto cost = cost_local[d_idx] = cost_ptr[idy * width * disp_range + idx * disp_range + d_idx];
			const auto cost= cost_ptr[idy * width * disp_range + idx * disp_range + d_idx];
			if (min_cost > cost) {
				min_cost = cost;
				best_disparity = d;
			}
		}

#if is_check_unique
		if (is_check_unique) {
			// 再遍历一次，输出次最小代价值
			for (sint32 d = min_disparity; d < max_disparity; d++) {
				if (d == best_disparity) {
					// 跳过最小代价值
					continue;
				}
				const auto& cost = cost_ptr[idy * width * disp_range + idx * disp_range + d];
				sec_min_cost = MIN(sec_min_cost, cost);
			}

			// 判断唯一性约束
			// 若(min-sec)/min < min*(1-uniquness)，则为无效估计
			if (sec_min_cost - min_cost <= static_cast<float32>(min_cost * (1 - uniqueness_ratio))) {
				disparity[idy * width + idx] = Invalid_Float;
				return;
			}
		}
#endif
		// ---子像素拟合
		if (best_disparity == min_disparity || best_disparity == max_disparity - 1) {
			disparity[idy * width + idx] = Invalid_Float;
			return;
		}
		// 最优视差前一个视差的代价值cost_1，后一个视差的代价值cost_2
		const sint32 idx_1 = best_disparity - 1 - min_disparity;
		const sint32 idx_2 = best_disparity + 1 - min_disparity;
		const float32 cost_1 = cost_ptr[idy * width * disp_range + idx * disp_range + idx_1];
		const float32 cost_2 = cost_ptr[idy * width * disp_range + idx * disp_range + idx_2]; 
		// 解一元二次曲线极值
		const float32 denom = cost_1 + cost_2 - 2 * min_cost;
		if (denom != 0.0f) {
			disparity[idy * width + idx] = static_cast<float32>(best_disparity) + (cost_1 - cost_2) / (denom * 2.0f);
		}
		else {
			disparity[idy * width + idx] = static_cast<float32>(best_disparity);
		}
		//free(cost_local);
	}

}



void ADCensusStereo::ComputeDisparity()
{
	const sint32& min_disparity = option_.min_disparity;
	const sint32& max_disparity = option_.max_disparity;
	const sint32 disp_range = max_disparity - min_disparity;
	if (disp_range <= 0) {
		return;
	}

	// 左影像视差图
	const auto disparity = disp_left_;
	// 左影像聚合代价数组
	const auto cost_ptr = aggregator_.get_cost_ptr();

	const sint32 width = width_;
	const sint32 height = height_;
	auto img_size = width * height;

	

	// ---逐像素计算最优视差
//#pragma omp parallel for 
//	for (sint32 i = 0; i < height; i++) {
//		// 为了加快读取效率，把单个像素的所有代价值存储到局部数组里
//		std::vector<float32> cost_local(disp_range);
//		for (sint32 j = 0; j < width; j++) {
//			float32 min_cost = Large_Float;
//			sint32 best_disparity = 0;
//
//			// ---遍历视差范围内的所有代价值，输出最小代价值及对应的视差值
//			for (sint32 d = min_disparity; d < max_disparity; d++) {
//				const sint32 d_idx = d - min_disparity;
//				const auto& cost = cost_local[d_idx] = cost_ptr[i * width * disp_range + j * disp_range + d_idx];
//				if (min_cost > cost) {
//					min_cost = cost;
//					best_disparity = d;
//				}
//			}
//			// ---子像素拟合
//			if (best_disparity == min_disparity || best_disparity == max_disparity - 1) {
//				disparity[i * width + j] = Invalid_Float;
//				continue;
//			}
//			// 最优视差前一个视差的代价值cost_1，后一个视差的代价值cost_2
//			const sint32 idx_1 = best_disparity - 1 - min_disparity;
//			const sint32 idx_2 = best_disparity + 1 - min_disparity;
//			const float32 cost_1 = cost_local[idx_1];
//			const float32 cost_2 = cost_local[idx_2];
//			// 解一元二次曲线极值
//			const float32 denom = cost_1 + cost_2 - 2 * min_cost;
//			if (denom != 0.0f) {
//				disparity[i * width + j] = static_cast<float32>(best_disparity) + (cost_1 - cost_2) / (denom * 2.0f);
//			}
//			else {
//				disparity[i * width + j] = static_cast<float32>(best_disparity);
//			}
//		}
//	}

	dim3 threadsPerBlock(32, 32);
	dim3 blocksPerGrid((width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
		(height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);

	float32* cost_ptr_cuda = nullptr;
	float32* disparity_cuda = nullptr;
	CUDA_CHECK_RETURN(hipMalloc((void**)&cost_ptr_cuda, sizeof(float32) * img_size * disp_range));
	CUDA_CHECK_RETURN(hipMalloc((void**)&disparity_cuda, sizeof(float32) * img_size));


	CUDA_CHECK_RETURN(hipMemcpyAsync(cost_ptr_cuda, cost_ptr, sizeof(float32) * img_size * disp_range, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpyAsync(disparity_cuda, disparity, sizeof(float32) * img_size, hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipDeviceSynchronize());



	// ---逐像素计算最优视差
	ComputeDisparityCuda << <blocksPerGrid, threadsPerBlock >> > (cost_ptr_cuda, disparity_cuda, height, width,
		max_disparity, min_disparity);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpyAsync(disparity, disparity_cuda, sizeof(float32) * img_size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	hipFree(cost_ptr_cuda);
	hipFree(disparity_cuda);

}

void ADCensusStereo::ComputeDisparityRight()
{
	const sint32& min_disparity = option_.min_disparity;
	const sint32& max_disparity = option_.max_disparity;
	const sint32 disp_range = max_disparity - min_disparity;
	if (disp_range <= 0) {
		return;
	}

	// 右影像视差图
	const auto disparity = disp_right_;
	// 左影像聚合代价数组
	const auto cost_ptr = aggregator_.get_cost_ptr();

	const sint32 width = width_;
	const sint32 height = height_;

	// 为了加快读取效率，把单个像素的所有代价值存储到局部数组里
	std::vector<float32> cost_local(disp_range);

	// ---逐像素计算最优视差
	// 通过左影像的代价，获取右影像的代价
	// 右cost(xr,yr,d) = 左cost(xr+d,yl,d)
#pragma omp parallel for 
	for (sint32 i = 0; i < height; i++) {
		for (sint32 j = 0; j < width; j++) {
			float32 min_cost = Large_Float;
			sint32 best_disparity = 0;

			// ---统计候选视差下的代价值
			for (sint32 d = min_disparity; d < max_disparity; d++) {
				const sint32 d_idx = d - min_disparity;
				const sint32 col_left = j + d;
				if (col_left >= 0 && col_left < width) {
					const auto& cost = cost_local[d_idx] = cost_ptr[i * width * disp_range + col_left * disp_range + d_idx];
					if (min_cost > cost) {
						min_cost = cost;
						best_disparity = d;
					}
				}
				else {
					cost_local[d_idx] = Large_Float;
				}
			}

			// ---子像素拟合
			if (best_disparity == min_disparity || best_disparity == max_disparity - 1) {
				disparity[i * width + j] = best_disparity;
				continue;
			}

			// 最优视差前一个视差的代价值cost_1，后一个视差的代价值cost_2
			const sint32 idx_1 = best_disparity - 1 - min_disparity;
			const sint32 idx_2 = best_disparity + 1 - min_disparity;
			const float32 cost_1 = cost_local[idx_1];
			const float32 cost_2 = cost_local[idx_2];
			// 解一元二次曲线极值
			const float32 denom = cost_1 + cost_2 - 2 * min_cost;
			if (denom != 0.0f) {
				disparity[i * width + j] = static_cast<float32>(best_disparity) + (cost_1 - cost_2) / (denom * 2.0f);
			}
			else {
				disparity[i * width + j] = static_cast<float32>(best_disparity);
			}
		}
	}
}

void ADCensusStereo::Release()
{
	SAFE_DELETE(disp_left_);
	SAFE_DELETE(disp_right_);
}

